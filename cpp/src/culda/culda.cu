// Copyright (c) 2021 Jisang Yoon
// All rights reserved.
//
// This source code is licensed under the Apache 2.0 license found in the
// LICENSE file in the root directory of this source tree.
#include "culda/culda.hpp"
#include "culda/cuda_lda_kernels.cuh"

namespace cusim {

CuLDA::CuLDA() {
  logger_ = CuSimLogger().get_logger();
  dev_info_ = GetDeviceInfo();
  if (dev_info_.unknown) DEBUG0("Unknown device type");
  INFO("cuda device info, major: {}, minor: {}, multi processors: {}, cores: {}",
       dev_info_.major, dev_info_.minor, dev_info_.mp_cnt, dev_info_.cores);
}

CuLDA::~CuLDA() {}

bool CuLDA::Init(std::string opt_path) {
  std::ifstream in(opt_path.c_str());
  if (not in.is_open()) return false;

  std::string str((std::istreambuf_iterator<char>(in)),
      std::istreambuf_iterator<char>());
  std::string err_cmt;
  auto _opt = json11::Json::parse(str, err_cmt);
  if (not err_cmt.empty()) return false;
  opt_ = _opt;
  CuSimLogger().set_log_level(opt_["c_log_level"].int_value());
  num_topics_ = opt_["num_topics"].int_value();
  block_dim_ = opt_["block_dim"].int_value();
  block_cnt_ = opt_["hyper_threads"].number_value() * (dev_info_.cores / block_dim_);
  INFO("num_topics: {}, block_dim: {}, block_cnt: {}", num_topics_, block_dim_, block_cnt_);
  return true;
}

void CuLDA::LoadModel(float* alpha, float* beta, int num_words) {
  num_words_ = num_words;
  DEBUG("copy model({} x {})", num_topics_, num_words_);
  dev_alpha_.resize(num_topics_);
  dev_beta_.resize(num_topics_ * num_words_);
  thrust::copy(alpha, alpha + num_topics_, dev_alpha_.begin());
  thrust::copy(beta, beta + num_topics_ * num_words_, dev_beta_.begin());
  alpha_ = alpha; beta_ = beta;
  InitModel();
}

void CuLDA::InitModel() {
  // resize device vector
  dev_grad_alpha_.resize(num_topics_);
  dev_new_beta_.resize(num_topics_ * num_words_);
  dev_gamma_.resize(num_topics_ * block_cnt_);
  dev_new_gamma_.resize(num_topics_ * block_cnt_);
  dev_phi_.resize(num_topics_ * block_cnt_);
  
  // resize host vector
  grad_alpha_.resize(num_topics_);
  new_beta_.resize(num_topics_ * num_words_);

  // fill zeros
  std::fill(grad_alpha_.begin(), grad_alpha_.end(), 0);
  std::fill(new_beta_.begin(), new_beta_.end(), 0);

  // copy to device
  thrust::copy(grad_alpha_.begin(), grad_alpha_.end(), dev_grad_alpha_.begin());
  thrust::copy(new_beta_.begin(), new_beta_.end(), dev_new_beta_.begin());
  CHECK_CUDA(hipDeviceSynchronize());
}

void CuLDA::FeedData(const int* indices, const int* indptr, 
    const int num_indices, const int num_indptr, const int num_iters) {
  thrust::device_vector<int> dev_indices(num_indices);
  thrust::device_vector<int> dev_indptr(num_indptr + 1);
  thrust::copy(indices, indices + num_indices, dev_indices.begin());
  thrust::copy(indptr, indptr + num_indptr + 1, dev_indptr.begin());
  CHECK_CUDA(hipDeviceSynchronize());

  EstepKernel<<<block_cnt_, block_dim_>>>(
    thrust::raw_pointer_cast(dev_indices.data()),
    thrust::raw_pointer_cast(dev_indptr.data()),
    num_indices, num_indptr, num_words_, num_topics_, num_iters,
    thrust::raw_pointer_cast(dev_gamma_.data()),
    thrust::raw_pointer_cast(dev_new_gamma_.data()),
    thrust::raw_pointer_cast(dev_phi_.data()),
    thrust::raw_pointer_cast(dev_alpha_.data()),
    thrust::raw_pointer_cast(dev_beta_.data()),
    thrust::raw_pointer_cast(dev_grad_alpha_.data()),
    thrust::raw_pointer_cast(dev_new_beta_.data()));
  
  CHECK_CUDA(hipDeviceSynchronize());
}

void CuLDA::Mstep(const int num_docs) {
  thrust::copy(dev_grad_alpha_.begin(), dev_grad_alpha_.end(), grad_alpha_.begin());
  thrust::copy(dev_new_beta_.begin(), dev_new_beta_.end(), new_beta_.begin());
  CHECK_CUDA(hipDeviceSynchronize());
  

}

} // namespace cusim
